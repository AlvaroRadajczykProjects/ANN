#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <cuda_pipeline.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <cassert>

#define N 256 * 8192 * 15
#define NITER 100

void imprimirVectorPorPantalla(char* texto_mostrar, float vector[], int inicio, int fin) {
    printf("\n%s [ ", texto_mostrar);
    for (int i = inicio; i < fin; i++) {
        printf("%.8f", vector[i]);
        if (i < fin - 1) { printf(","); }
        printf(" ");
    }
    printf("]");
}

using func_t = float(*) (float);

__global__ void applyFunctionScalar(float* arr) {
    arr[blockIdx.x * blockDim.x + threadIdx.x] = 3*arr[blockIdx.x * blockDim.x + threadIdx.x];
}

//no provoca que se haga más lenta la operación en absoluto!
__device__ float funcion_watelu(float x) {
    //if (x < 0) { return expf(x); }
    //else { return x; }
    return x*12345;
}
__device__ func_t p_add_func = funcion_watelu;

__device__ float funcion_wate(float x) {
    //if (x < 0) { return expf(x); }
    //else { return x; }
    return x * 54321;
}
__device__ func_t p2_add_func = funcion_wate;

__global__ void prueba(func_t* lfunc) {
    lfunc[0] = funcion_watelu;
    lfunc[1] = funcion_wate;
}

//https://forums.developer.nvidia.com/t/the-float-and-float4-types-in-cuda/65061
__global__ void applyFunctionVectorial(float* arr, func_t func) {
    float4 val = reinterpret_cast<float4*>(arr)[blockIdx.x * blockDim.x + threadIdx.x];
    val.x = func(val.x);
    val.y = func(val.y);
    val.z = func(val.z);
    val.w = func(val.w);
    reinterpret_cast<float4*>(arr)[blockIdx.x * blockDim.x + threadIdx.x] = val;
}

void xd(float* p, func_t func) {
    //func_t h_add_func;
    //hipMemcpyFromSymbol(&h_add_func, HIP_SYMBOL(&symbol), sizeof(func_t));
    printf("\nLo estoy haciendo en la funcion!");
    applyFunctionVectorial << < (int)ceil(N / (float)(1024 * 4)), 1024 >> > (p, func);
}

float caca(float x) {
    //if (x < 0) { return expf(x); }
    //else { return x; }
    return x * 12345;
}

func_t getDeviceSymbolInGlobalMemory(func_t d_arrfunc) {
    func_t h_arrfunc;
    hipMemcpy(&h_arrfunc, d_arrfunc, sizeof(func_t), hipMemcpyDeviceToHost);
    return h_arrfunc;
}

func_t d_arrfunc = 0;

int main() {

    hipError_t cudaStatus;

    hipMalloc(&d_arrfunc, sizeof(func_t));

    /*cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\nError A: %s", hipGetErrorString(cudaStatus));
        return 0;
    }*/

    //func_t h_add_func;
    //hipMemcpyFromSymbol(&h_add_func, HIP_SYMBOL(p_add_func), sizeof(func_t));

    float* p = 0;
    hipMalloc((void**)&p, N * sizeof(float));

    float* h_p = new float[N];
    for (int i = 0; i < N; i++) { h_p[i] = 100; }

    //imprimirVectorPorPantalla("antes", h_p, 0, N);

    hipMemcpy(p, h_p, N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time;

    hipGetSymbolAddress((void**)&d_arrfunc, p_add_func);
    func_t h_arrfunc = getDeviceSymbolInGlobalMemory(d_arrfunc);

    hipGetSymbolAddress((void**)&d_arrfunc, p2_add_func);
    func_t h_arrfunc2 = getDeviceSymbolInGlobalMemory(d_arrfunc);
    
    
    //printf("\nf1: %p", *h_arrfunc);

    //applyFunctionVectorial << < (int)ceil(N / (float)(1024 * 4)), 1024 >> > (p, h_arrfunc2);
    xd(p, h_arrfunc2);

    /*func_t* h_arrfunc = new func_t[2];
    func_t* d_arrfunc = 0;
    hipMalloc(&d_arrfunc, 2 * sizeof(func_t));
    prueba << < 1, 1 >> > (d_arrfunc);
    hipMemcpy(h_arrfunc, d_arrfunc, 2 * sizeof(func_t*), hipMemcpyDeviceToHost);*/

    //hipMemcpyFromSymbol(&h_arrfunc, HIP_SYMBOL(lfunc), 2*sizeof(func_t*));

    //printf("\nprueba: %p", h_arrfunc[0]);
    //printf("\nprueba: %p", h_arrfunc[1]);
    //xd(p, h_arrfunc);

    

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    //applyFunctionVectorial << < (int)ceil(N / (float)(1024 * 4)), 1024 >> > (p, h_arrfunc[1]);

    hipDeviceSynchronize();

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\nError Kernel: %s", hipGetErrorString(cudaStatus));
        return 0;
    }

    /*
    //execute kernel
    for (int i = 0; i < NITER; i++) {
        //applyFunctionScalar << < (int)ceil(N / (float)1024), 1024 >> > (p);
        //applyFunctionVectorial << < (int)ceil(N / (float)(1024 * 4)), 1024 >> > (p);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "\nError");
            return 0;
        }
    }*/

    //xd(p, (const void* ) p_add_func);
    //xd(p, (const void*)funcion_watelu); //???
    //applyFunctionScalar << < (int)ceil(N / (float)1024), 1024 >> > (h_add_func, p);

    //applyFunctionVectorial << < (int)ceil(N / (float)(1024 * 4)), 1024 >> > (p, h_add_func);

    
    //hipGetSymbolAddress((void**)&temp, p_add_func);

    //hipGetSymbolAddress((void**)&temp, p_add_func);

    /*cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\nError 1: %s", hipGetErrorString(cudaStatus));
        return 0;
    }

    

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "\nError 2: %s", hipGetErrorString(cudaStatus));
        return 0;
    }*/
    
    //hipMemcpyFromSymbol(&h_add_func, HIP_SYMBOL("p_add_func"), sizeof(func_t));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("\n\nExecution time: %f ms\n\n", time/(float)NITER);

    //hipMemcpy(p, h_p, N * sizeof(float), hipMemcpyHostToDevice);

    //applyFunctionScalar << < (int)ceil(N / (float)1024), 1024 >> > (p);
    //applyFunctionVectorial << < (int)ceil(N / (float)(1024*4)), 1024 >> > (p);

    hipMemcpy(h_p, p, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    imprimirVectorPorPantalla("despues", h_p, 0, 20);

    hipFree(p);
    free(h_p);

    return 0;
}

/*

// ReLU kernel using float4 data type for improved memory access
__global__ void reluKernel(const float* input, float* output, int numElements) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float inputValue = input[tid];
    output[tid] = inputValue;
    
}

int main() {
    int numElements = 1024*8192;  // Number of elements in the array
    size_t arraySize = numElements * sizeof(float);

    float* h_input, * h_output;  // Host arrays
    float* d_input, * d_output;  // Device arrays

    // Allocate and initialize host input array h_input

    // Allocate host output array
    h_output = (float*)malloc(arraySize);
    h_input = (float*)malloc(arraySize);

    // Allocate device memory
    hipMalloc((void**)&d_input, arraySize);
    hipMalloc((void**)&d_output, arraySize);

    // Copy input data from host to device
    hipMemcpy(d_input, h_input, arraySize, hipMemcpyHostToDevice);

    // Launch the kernel
    int blockSize = 1024;
    int numBlocks = 8192;

    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    reluKernel << <numBlocks, blockSize >> > (d_input, d_output, numElements);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("\n\nExecution time: %f ms\n\n", time / (float)NITER);

    // Copy the result from device to host
    hipMemcpy(h_output, d_output, arraySize, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    free(h_input);
    free(h_output);

    return 0;
}

*/