#include "hip/hip_runtime.h"
#include "Network.cuh"

using namespace std;

Network::Network(int is, int nn, int nl, Layer** ls) {
	input_size = is;
	output_size = ls[nl - 1]->getSize();
	number_networks = nn;
	number_layers = nl;
	layers = ls;
	layers[0]->setIsFirstLayer(true);
	hipblasCreate(&handle);
	for (int i = 0; i < number_layers; i++) {
		max_layer_size = max(max_layer_size, layers[i]->getSize());
		layers[i]->setCublasHandle(&handle);
		layers[i]->setNumberNetworks(number_networks);
		layers[i]->setIsTraining(false);
		if (i == 0) { layers[i]->setInputSize(input_size); }
		else { layers[i]->setInputSize(layers[i-1]->getSize()); }
		layers[i]->allocWeightMatricesMemory();
	}
	hipDeviceSynchronize();
}

Network::~Network() {
	for (int i = 0; i < number_layers; i++) {
		layers[i]->freeWeightMatricesMemory();
		delete layers[i];
	}
	delete layers;
	hipblasDestroy(handle);
	hipDeviceSynchronize();
}

void Network::showInfoAboutNetwork() {
	printf("\n");
	printf("\nINFO ABOUT THE NETWORK");
	printf("\n======================");
	printf("\nInput size (number of each example attributes): %d", input_size);
	//printf("\nMax input examples (this network can forward training and/or predicting): %d", input_size);
	printf("\nOutput size (number of each prediction): %d", output_size);
	printf("\nNumber of networks (multiple networks can be trained for ensemble averaging with multiple similar neural networks in one device): %d", number_networks);
	printf("\nNumber of layers (all networks are similar, same shape, different initialization values): %d", number_layers);
	printf("\nMax layer size: %d", max_layer_size);
	printf("\nLayers dimensions:");
	for (int i = 0; i < number_layers; i++) {
		printf("\n\tLayer %d:", i);
		layers[i]->showInfo();
	}
	printf("\n");
}

void Network::showWeightsBiasesLayers() {
	printf("\n");
	printf("\nWEIGHTS AND BIASES");
	printf("\n==================");
	for (int i = 0; i < number_layers; i++) {
		printf("\nLayer %d:",i);
		layers[i]->showWeightBias();
	}
	printf("\n");
}

void Network::showAuxiliarExpandReduceMatrices() {
	printf("\n");
	printf("\nAUXILIAR EXPAND AND REDUCE VECTORS (is only one, but check all networks match the same)");
	printf("\n==================================------------------------------------------------------");
	printf("\n");
	for (int i = 0; i < number_layers; i++) {
		printf("\nLayer %d:", i);
		layers[i]->showAuxiliarExpandReduce();
	}
	printf("\n");
}

void Network::showForwardMatrices() {
	printf("\n");
	printf("\nFORWARD MATRICES");
	printf("\n================");
	for (int i = 0; i < number_layers; i++) {
		printf("\nLayer %d:", i);
		layers[i]->showForward();
	}
	printf("\n");
}

void Network::initForward(int max_num_input_examples_expected) {
	max_input_number_examples = max_num_input_examples_expected;
	d_pinned_output_offset = input_size * max_input_number_examples;
	hipStreamCreate(&stream_principal);
	hipStreamCreate(&stream_transferencia_output);
	hipblasSetStream(handle, stream_principal);
	hipHostAlloc(&h_pinned_input_matrix, input_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	hipHostAlloc(&h_pinned_output_matrix, output_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	hipMalloc(&d_pinned_input_output_auxiliar_matrix, max_input_number_examples * ( input_size + output_size) * sizeof(float));
	
	hipMalloc(&d_auxiliar_expand_reduce_matrix, max_num_input_examples_expected * sizeof(float));
	float* h_auxiliar_expand_reduce_matrix = new float[max_num_input_examples_expected];
	for (int i = 0; i < max_num_input_examples_expected; i++) { h_auxiliar_expand_reduce_matrix[i] = 1.0f; }
	hipMemcpy(d_auxiliar_expand_reduce_matrix, h_auxiliar_expand_reduce_matrix, max_num_input_examples_expected * sizeof(float), hipMemcpyHostToDevice);
	delete h_auxiliar_expand_reduce_matrix;
	for (int i = 0; i < number_layers; i++) {
		layers[i]->setNumberInputExamples(max_input_number_examples);
		layers[i]->setAuxiliarExpandReduceMatrix(d_auxiliar_expand_reduce_matrix);
		layers[i]->allocForwardMemory();
	}
	hipDeviceSynchronize();
}

const void Network::forward(int num_examples, float* input_data, float* output_pointer_dest) {
	if (num_examples <= max_input_number_examples) {
		hipMemcpyAsync(h_pinned_input_matrix, input_data, num_examples * input_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix, h_pinned_input_matrix, num_examples * input_size * sizeof(float), hipMemcpyHostToDevice, stream_principal);
		layers[0]->forward(d_pinned_input_output_auxiliar_matrix);
		for (int i = 1; i < number_layers; i++) {
			layers[i]->forward(layers[i-1]);
		}
		hipStreamSynchronize(stream_principal);
	} else {
		printf("\nCannot make forward, more examples than max number of examples defined in initForward");
	}
}

const void Network::forwardTrain(int num_examples, float* input_data, float* output_data) {
	if (num_examples <= max_input_number_examples) {
		hipMemcpyAsync(h_pinned_input_matrix, input_data, num_examples * input_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
		hipMemcpyAsync(h_pinned_output_matrix, output_data, num_examples * output_size * sizeof(float), hipMemcpyHostToHost, stream_transferencia_output);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix, h_pinned_input_matrix, num_examples * input_size * sizeof(float), hipMemcpyHostToDevice, stream_principal);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix + d_pinned_output_offset, h_pinned_output_matrix, num_examples * output_size * sizeof(float), hipMemcpyHostToDevice, stream_transferencia_output);
		//para hacer el backward, esperar� a que ambas transferencias hayan terminado
		hipStreamSynchronize(stream_principal);
		hipStreamSynchronize(stream_transferencia_output);
	}
	else {
		printf("\nCannot make forward, more examples than max number of examples defined in initForward");
	}
}

void Network::finalizeForward() {
	hipblasSetStream(handle, 0);
	hipStreamDestroy(stream_principal);
	hipStreamDestroy(stream_transferencia_output);
	hipFree(d_pinned_input_output_auxiliar_matrix);
	hipHostFree(h_pinned_input_matrix);
	hipHostFree(h_pinned_output_matrix);
	
	for (int i = 0; i < number_layers; i++) {
		layers[i]->setNumberInputExamples(0);
		layers[i]->setAuxiliarExpandReduceMatrix(NULL);
		layers[i]->freeForwardMemory();
	}
	hipFree(d_auxiliar_expand_reduce_matrix);
	hipDeviceSynchronize();
}