#include "hip/hip_runtime.h"
#include "Network.cuh"

using namespace std;

Network::Network(int is, int nn, int nl, Layer** ls) {
	input_size = is;
	output_size = ls[nl - 1]->getSize();
	number_networks = nn;
	number_layers = nl;
	layers = ls;
	hipblasCreate(&handle);
	for (int i = 0; i < number_layers; i++) {
		max_layer_size = max(max_layer_size, layers[i]->getSize());
		layers[i]->setCublasHandle(&handle);
		layers[i]->setNumberNetworks(number_networks);
		if (i == 0) { layers[i]->setInputSize(input_size); }
		else { layers[i]->setInputSize(layers[i-1]->getSize()); }
		layers[i]->allocMemory();
	}
	hipDeviceSynchronize();
}

Network::~Network() {
	for (int i = 0; i < number_layers; i++) {
		layers[i]->freeMemory();
		delete layers[i];
	}
	delete layers;
	hipblasDestroy(handle);
	hipDeviceSynchronize();
}

void Network::showInfoAboutNetwork() {
	printf("\nInput size (number of each example attributes): %d", input_size);
	//printf("\nMax input examples (this network can forward training and/or predicting): %d", input_size);
	printf("\nOutput size (number of each prediction): %d", output_size);
	printf("\nNumber of networks (multiple networks can be trained for ensemble averaging with multiple similar neural networks in one device): %d", number_networks);
	printf("\nNumber of layers (all networks are similar, same shape, different initialization values): %d", number_layers);
	printf("\nMax layer size: %d", max_layer_size);
	printf("\nLayers dimensions:");
	for (int i = 0; i < number_layers; i++) {
		printf("\n\tLayer %d:", i);
		layers[i]->showInfo();
	}
	printf("\n");
}

void Network::showWeightsBiasesLayers() {
	printf("\n");
	for (int i = 0; i < number_layers; i++) {
		layers[i]->showWeightBias();
	}
	printf("\n");
}

void Network::initForward(int max_num_input_examples_expected) {
	max_input_number_examples = max_num_input_examples_expected;
	hipHostAlloc(&h_pinned_input_matrix, input_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	hipHostAlloc(&h_pinned_output_matrix, output_size * max_input_number_examples * sizeof(float), 0);
	d_pinned_output_offset = input_size * max_input_number_examples;
	hipMalloc(&d_pinned_input_output_auxiliar_matrix, max_input_number_examples * (input_size + output_size) * sizeof(float));
	hipDeviceSynchronize();
}

const void Network::forward(int num_examples, float* input_data, float* output_pointer_dest) {
	if (num_examples <= max_input_number_examples) {
		hipMemcpyAsync(h_pinned_input_matrix, input_data, num_examples * input_size * sizeof(float), hipMemcpyHostToHost);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix, h_pinned_input_matrix, num_examples * input_size * sizeof(float), hipMemcpyHostToDevice);
	} else {
		printf("\nCannot make forward, more examples than max number of examples defined in initForward");
	}
}

void Network::finalizeForward() {
	hipFree(d_pinned_input_output_auxiliar_matrix);
	hipHostFree(h_pinned_input_matrix);
	hipHostFree(h_pinned_output_matrix);
	hipDeviceSynchronize();
}

void Network::initForwardTrain(int max_num_input_examples_expected) {
	hipblasCreate(&handle);
	max_input_number_examples = max_num_input_examples_expected;
	hipStreamCreate(&stream_principal);
	hipStreamCreate(&stream_transferencia_output);
	hipHostAlloc(&h_pinned_input_matrix, input_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	hipHostAlloc(&h_pinned_output_matrix, output_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	d_pinned_output_offset = input_size * max_input_number_examples;
	hipMalloc(&d_pinned_input_output_auxiliar_matrix, max_input_number_examples * max(number_networks * max_layer_size, input_size + output_size) * sizeof(float));
	hipDeviceSynchronize();
}

const void Network::forwardTrain(int num_examples, float* input_data, float* output_data) {
	if (num_examples <= max_input_number_examples) {
		hipMemcpyAsync(h_pinned_input_matrix, input_data, num_examples * input_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
		hipMemcpyAsync(h_pinned_output_matrix, output_data, num_examples * output_size * sizeof(float), hipMemcpyHostToHost, stream_transferencia_output);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix, h_pinned_input_matrix, num_examples * input_size * sizeof(float), hipMemcpyHostToDevice, stream_principal);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix + d_pinned_output_offset, h_pinned_output_matrix, num_examples * output_size * sizeof(float), hipMemcpyHostToDevice, stream_transferencia_output);
	}
	else {
		printf("\nCannot make forward, more examples than max number of examples defined in initForward");
	}
}

void Network::finalizeForwardTrain() {
	hipStreamDestroy(stream_principal);
	hipStreamDestroy(stream_transferencia_output);
	hipFree(d_pinned_input_output_auxiliar_matrix);
	hipHostFree(h_pinned_input_matrix);
	hipHostFree(h_pinned_output_matrix);
	hipDeviceSynchronize();
}