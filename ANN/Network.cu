#include "hip/hip_runtime.h"
#include "Network.cuh"

using namespace std;

Network::Network(int is, int nn, int nl, Layer** ls) {
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	max_num_threads = deviceProp.maxThreadsPerBlock;

	input_size = is;
	output_size = ls[nl - 1]->getSize();
	number_networks = nn;
	number_layers = nl;
	layers = ls;
	layers[0]->setIsFirstLayer(true);
	hipblasCreate(&handle);
	for (int i = 0; i < number_layers; i++) {
		max_layer_size = max(max_layer_size, layers[i]->getSize());
		layers[i]->setCublasHandle(&handle);
		layers[i]->setNumberNetworks(number_networks);
		layers[i]->setIsTraining(false);
		if (i == 0) { layers[i]->setInputSize(input_size); }
		else { layers[i]->setInputSize(layers[i-1]->getSize()); }
		layers[i]->setMaxNumThreads(max_num_threads);
		layers[i]->allocWeightMatricesMemory();
	}
	hipDeviceSynchronize();
}

Network::~Network() {
	for (int i = 0; i < number_layers; i++) {
		layers[i]->freeWeightMatricesMemory();
		delete layers[i];
	}
	delete layers;
	hipblasDestroy(handle);
	hipDeviceSynchronize();
}

void Network::showInfoAboutNetwork() {
	printf("\n");
	printf("\nINFO ABOUT THE NETWORK");
	printf("\n======================");
	printf("\nInput size (number of each example attributes): %d", input_size);
	//printf("\nMax input examples (this network can forward training and/or predicting): %d", input_size);
	printf("\nOutput size (number of each prediction): %d", output_size);
	printf("\nNumber of networks (multiple networks can be trained for ensemble averaging with multiple similar neural networks in one device): %d", number_networks);
	printf("\nNumber of layers (all networks are similar, same shape, different initialization values): %d", number_layers);
	printf("\nMax layer size: %d", max_layer_size);
	printf("\nLayers dimensions:");
	for (int i = 0; i < number_layers; i++) {
		printf("\n\tLayer %d:", i);
		layers[i]->showInfo();
	}
	printf("\n");
}

void Network::showWeightsBiasesLayers() {
	printf("\n");
	printf("\nWEIGHTS AND BIASES");
	printf("\n==================");
	for (int i = 0; i < number_layers; i++) {
		printf("\nLayer %d:",i);
		layers[i]->showWeightBias();
	}
	printf("\n");
}

void Network::showAuxiliarExpandReduceMatrices() {
	printf("\n");
	printf("\nAUXILIAR EXPAND AND REDUCE VECTORS (is only one, but check all networks match the same)");
	printf("\n==================================------------------------------------------------------");
	printf("\n");
	for (int i = 0; i < number_layers; i++) {
		printf("\nLayer %d:", i);
		layers[i]->showAuxiliarExpandReduce();
	}
	printf("\n");
}

void Network::showForwardMatrices() {
	printf("\n");
	printf("\nFORWARD MATRICES");
	printf("\n================");
	for (int i = 0; i < number_layers; i++) {
		printf("\nLayer %d:", i);
		layers[i]->showForward();
	}
	printf("\n");
}

void Network::initForward(int max_num_input_examples_expected) {
	max_input_number_examples = max_num_input_examples_expected;
	d_pinned_output_offset = input_size * max_input_number_examples;
	hipStreamCreate(&stream_principal);
	hipStreamCreate(&stream_transferencia_output);
	hipblasSetStream(handle, stream_principal);
	hipHostAlloc(&h_pinned_input_matrix, input_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	hipHostAlloc(&h_pinned_output_matrix, output_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	hipMalloc(&d_pinned_input_output_auxiliar_matrix, max_input_number_examples * ( input_size + output_size) * sizeof(float));

	float** hd_input_pointers = new float* [number_networks];
	for (int i = 0; i < number_networks; i++) { hd_input_pointers[i] = d_pinned_input_output_auxiliar_matrix + 0; }
	hipMalloc(&d_input_pointers, number_networks*sizeof(float*));
	hipMemcpy(d_input_pointers, hd_input_pointers, number_networks * sizeof(float*), hipMemcpyHostToDevice);
	delete hd_input_pointers;
	
	int tam = max(max_input_number_examples, number_networks);
	hipMalloc(&d_auxiliar_expand_reduce_matrix, tam * sizeof(float));
	float* h_auxiliar_expand_reduce_matrix = new float[tam];
	for (int i = 0; i < tam; i++) { h_auxiliar_expand_reduce_matrix[i] = 1.0f; }
	hipMemcpy(d_auxiliar_expand_reduce_matrix, h_auxiliar_expand_reduce_matrix, tam * sizeof(float), hipMemcpyHostToDevice);
	delete h_auxiliar_expand_reduce_matrix;
	for (int i = 0; i < number_layers; i++) {
		layers[i]->setNumberInputExamples(max_input_number_examples);
		layers[i]->setAuxiliarExpandReduceMatrix(d_auxiliar_expand_reduce_matrix);
		layers[i]->allocForwardMemory();
	}

	hipMalloc(&d_output_forward_multiple_nn_sum, nextFourMultiple(max_input_number_examples * output_size) * sizeof(float));

	hipDeviceSynchronize();
}

const void Network::forward(int num_examples, float* input_data, float* output_pointer_dest) {
	if (num_examples <= max_input_number_examples) {
		hipMemcpyAsync(h_pinned_input_matrix, input_data, num_examples * input_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix, h_pinned_input_matrix, num_examples * input_size * sizeof(float), hipMemcpyHostToDevice, stream_principal);
		layers[0]->forward(stream_principal, d_input_pointers);
		for (int i = 1; i < number_layers; i++) {
			layers[i]->forward(stream_principal, layers[i-1]);
		}

		if (number_networks == 1) {
			hipMemcpyAsync(h_pinned_output_matrix, layers[number_layers-1]->getDeviceForward(), num_examples * output_size * sizeof(float), hipMemcpyDeviceToHost, stream_principal);
			hipMemcpyAsync(output_pointer_dest, h_pinned_output_matrix, num_examples * output_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
		} else {
			if (max_input_number_examples == 1) {
				productoMatricesDevice(handle, d_auxiliar_expand_reduce_matrix, layers[number_layers - 1]->getDeviceForward(), d_output_forward_multiple_nn_sum, 1, number_networks, output_size);
				multiplyAllElementsByConstant << < (int)ceil(nextFourMultiple(num_examples * output_size) /(float)(max_num_threads*4)), min(max_num_threads, nextFourMultiple(num_examples * output_size) / 4), 0, stream_principal >> > (d_output_forward_multiple_nn_sum, 1 / (float)number_networks);
				hipMemcpyAsync(h_pinned_output_matrix, d_output_forward_multiple_nn_sum, num_examples * output_size * sizeof(float), hipMemcpyDeviceToHost, stream_principal);
				hipMemcpyAsync(output_pointer_dest, h_pinned_output_matrix, num_examples * output_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
			} else {
				//habr� que hacer el sumatorio de todas las matrices al de todas las redes, y multiplicarles 1/numero_redes
			}
		}

		hipStreamSynchronize(stream_principal);
	} else {
		printf("\nCannot make forward, more examples than max number of examples defined in initForward");
	}
}

const void Network::forwardTrain(int num_examples, float* input_data, float* output_data) {
	if (num_examples <= max_input_number_examples) {
		hipMemcpyAsync(h_pinned_input_matrix, input_data, num_examples * input_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
		hipMemcpyAsync(h_pinned_output_matrix, output_data, num_examples * output_size * sizeof(float), hipMemcpyHostToHost, stream_transferencia_output);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix, h_pinned_input_matrix, num_examples * input_size * sizeof(float), hipMemcpyHostToDevice, stream_principal);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix + d_pinned_output_offset, h_pinned_output_matrix, num_examples * output_size * sizeof(float), hipMemcpyHostToDevice, stream_transferencia_output);
		//para hacer el backward, esperar� a que ambas transferencias hayan terminado
		hipStreamSynchronize(stream_principal);
		hipStreamSynchronize(stream_transferencia_output);
	}
	else {
		printf("\nCannot make forward, more examples than max number of examples defined in initForward");
	}
}

void Network::finalizeForward() {
	hipblasSetStream(handle, 0);
	hipStreamDestroy(stream_principal);
	hipStreamDestroy(stream_transferencia_output);
	hipFree(d_pinned_input_output_auxiliar_matrix);
	hipFree(d_input_pointers);
	hipHostFree(h_pinned_input_matrix);
	hipHostFree(h_pinned_output_matrix);
	
	for (int i = 0; i < number_layers; i++) {
		layers[i]->setNumberInputExamples(0);
		layers[i]->setAuxiliarExpandReduceMatrix(NULL);
		layers[i]->freeForwardMemory();
	}
	hipFree(d_auxiliar_expand_reduce_matrix);

	hipFree(d_output_forward_multiple_nn_sum);

	hipDeviceSynchronize();
}