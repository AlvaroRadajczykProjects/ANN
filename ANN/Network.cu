#include "hip/hip_runtime.h"
#include "Network.cuh"

using namespace std;

Network::Network(int is, int nn, int nl, Layer** ls, func2_t ls_fn, func2_t dls_fn) {
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	max_num_threads = deviceProp.maxThreadsPerBlock;

	loss_function = ls_fn;
	derivative_loss_function = dls_fn;
	input_size = is;
	output_size = ls[nl - 1]->getSize();
	number_networks = nn;
	number_layers = nl;
	layers = ls;
	layers[0]->setIsFirstLayer(true);
	hipblasCreate(&handle);
	for (int i = 0; i < number_layers; i++) {
		max_layer_size = max(max_layer_size, layers[i]->getSize());
		layers[i]->setCublasHandle(&handle);
		layers[i]->setNumberNetworks(number_networks);
		layers[i]->setIsTraining(false);
		if (i == 0) { layers[i]->setInputSize(input_size); }
		else { layers[i]->setInputSize(layers[i-1]->getSize()); }
		layers[i]->setMaxNumThreads(max_num_threads);
		layers[i]->allocWeightMatricesMemory();
	}
	hipDeviceSynchronize();
}

Network::~Network() {
	for (int i = 0; i < number_layers; i++) {
		layers[i]->freeWeightMatricesMemory();
		delete layers[i];
	}
	delete layers;
	hipblasDestroy(handle);
	hipDeviceSynchronize();
}

void Network::showInfoAboutNetwork() {
	printf("\n");
	printf("\nINFO ABOUT THE NETWORK");
	printf("\n======================");
	printf("\nInput size (number of each example attributes): %d", input_size);
	//printf("\nMax input examples (this network can forward training and/or predicting): %d", input_size);
	printf("\nOutput size (number of each prediction): %d", output_size);
	printf("\nNumber of networks (multiple networks can be trained for ensemble averaging with multiple similar neural networks in one device): %d", number_networks);
	printf("\nNumber of layers (all networks are similar, same shape, different initialization values): %d", number_layers);
	printf("\nMax layer size: %d", max_layer_size);
	printf("\nLayers dimensions:");
	for (int i = 0; i < number_layers; i++) {
		printf("\n\tLayer %d:", i);
		layers[i]->showInfo();
	}
	printf("\n");
}

void Network::showWeightsBiasesLayers() {
	printf("\n");
	printf("\nWEIGHTS AND BIASES");
	printf("\n==================");
	for (int i = 0; i < number_layers; i++) {
		printf("\nLayer %d:",i);
		layers[i]->showWeightBias();
	}
	printf("\n");
}

void Network::showErrorWeightsBiasesLayers() {
	printf("\n");
	printf("\nERROR WEIGHTS AND BIASES");
	printf("\n========================");
	for (int i = 0; i < number_layers; i++) {
		printf("\nLayer %d:", i);
		layers[i]->showErrorWeightBias();
	}
	printf("\n");
}

void Network::showAuxiliarExpandReduceMatrices() {
	printf("\n");
	printf("\nAUXILIAR EXPAND AND REDUCE VECTORS (is only one, but check all networks match the same)");
	printf("\n==================================------------------------------------------------------");
	printf("\n");
	for (int i = 0; i < number_layers; i++) {
		printf("\nLayer %d:", i);
		layers[i]->showAuxiliarExpandReduce();
	}
	printf("\n");
}

void Network::showForwardMatrices() {
	printf("\n");
	printf("\nFORWARD MATRICES");
	printf("\n================");
	for (int i = 0; i < number_layers; i++) {
		printf("\nLayer %d:", i);
		layers[i]->showForward();
	}
	printf("\n");
}

int Network::getNumberNetwors() {
	return number_networks;
}

void Network::initForward(int max_num_input_examples_expected) {
	max_input_number_examples = max_num_input_examples_expected;
	d_pinned_output_offset = input_size * max_input_number_examples;
	hipStreamCreate(&stream_principal);
	hipStreamCreate(&stream_transferencia_output);
	hipblasSetStream(handle, stream_principal);
	hipHostAlloc(&h_pinned_input_matrix, input_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	hipHostAlloc(&h_pinned_output_matrix, output_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	hipMalloc(&d_pinned_input_output_auxiliar_matrix, max_input_number_examples * ( input_size + output_size) * sizeof(float));

	float** hd_input_pointers = new float* [number_networks];
	for (int i = 0; i < number_networks; i++) { hd_input_pointers[i] = d_pinned_input_output_auxiliar_matrix + 0; }
	hipMalloc(&d_input_pointers, number_networks*sizeof(float*));
	hipMemcpy(d_input_pointers, hd_input_pointers, number_networks * sizeof(float*), hipMemcpyHostToDevice);
	delete hd_input_pointers;
	
	int tam = nextFourMultiple(max(max(max_batch_size, number_networks), output_size));
	hipMalloc(&d_auxiliar_expand_reduce_matrix, tam * sizeof(float));
	float* h_auxiliar_expand_reduce_matrix = new float[tam];
	for (int i = 0; i < tam; i++) { h_auxiliar_expand_reduce_matrix[i] = 1.0f; }
	hipMemcpy(d_auxiliar_expand_reduce_matrix, h_auxiliar_expand_reduce_matrix, tam * sizeof(float), hipMemcpyHostToDevice);
	delete h_auxiliar_expand_reduce_matrix;
	for (int i = 0; i < number_layers; i++) {
		layers[i]->setNumberInputExamples(max_input_number_examples);
		layers[i]->setAuxiliarExpandReduceMatrix(d_auxiliar_expand_reduce_matrix);
		layers[i]->allocForwardMemory();
	}

	hipMalloc(&d_output_forward_multiple_nn_sum, nextFourMultiple(number_networks * max_input_number_examples * output_size) * sizeof(float));

	//Cublas warmup
	productoMatricesDevice(handle, d_auxiliar_expand_reduce_matrix, layers[number_layers - 1]->getDeviceForward(), d_output_forward_multiple_nn_sum, 1, number_networks, output_size);

	hipDeviceSynchronize();
}

void Network::initForwardTrain(int m_num_examples, int m_batch_size) {
	max_batch_size = m_batch_size;
	max_input_number_examples = m_num_examples;
	d_pinned_output_offset = input_size * max_input_number_examples;
	hipStreamCreate(&stream_principal);
	hipStreamCreate(&stream_transferencia_output);
	hipblasSetStream(handle, stream_principal);
	hipHostAlloc(&h_pinned_input_matrix, input_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	hipHostAlloc(&h_pinned_output_matrix, output_size * max_input_number_examples * sizeof(float), hipHostMallocWriteCombined);
	hipMalloc(&d_pinned_input_output_auxiliar_matrix, ( d_pinned_output_offset + nextFourMultiple(max_input_number_examples * output_size) ) * sizeof(float));

	float** hd_input_pointers = new float* [number_networks];
	for (int i = 0; i < number_networks; i++) { hd_input_pointers[i] = d_pinned_input_output_auxiliar_matrix + 0; }
	hipMalloc(&d_input_pointers, number_networks * sizeof(float*));
	hipMemcpy(d_input_pointers, hd_input_pointers, number_networks * sizeof(float*), hipMemcpyHostToDevice);
	delete hd_input_pointers;

	int tam = nextFourMultiple( max_batch_size * number_networks * output_size );
	hipMalloc(&d_auxiliar_expand_reduce_matrix, tam * sizeof(float));
	float* h_auxiliar_expand_reduce_matrix = new float[tam];
	for (int i = 0; i < tam; i++) { h_auxiliar_expand_reduce_matrix[i] = 1.0f; }
	hipMemcpy(d_auxiliar_expand_reduce_matrix, h_auxiliar_expand_reduce_matrix, tam * sizeof(float), hipMemcpyHostToDevice);
	delete h_auxiliar_expand_reduce_matrix;

	//max_input_number_examples instead of max_batch_size * number_networks?
	hipMalloc(&d_auxiliar_matrix_loss_function_error_backprop, nextFourMultiple(max_batch_size * number_networks * max_layer_size) * sizeof(float));
	hipMalloc(&d_auxiliar2_matrix_loss_function_error_backprop, nextFourMultiple(max_batch_size * number_networks * max_layer_size) * sizeof(float));

	for (int i = 0; i < number_layers; i++) {
		layers[i]->setNumberInputExamples(max_batch_size);
		layers[i]->setAuxiliarExpandReduceMatrix(d_auxiliar_expand_reduce_matrix);
		layers[i]->allocForwardMemory();
		layers[i]->allocBackwardMemory(m_batch_size, d_auxiliar_matrix_loss_function_error_backprop, d_auxiliar2_matrix_loss_function_error_backprop);
		layers[i]->setIsTraining(true);
	}

	hipMalloc(&d_output_forward_multiple_nn_sum, nextFourMultiple(number_networks * max_batch_size * output_size) * sizeof(float));
	float** hd_output_forward_multiple_nn_sum_pointers = new float* [number_networks];
	hipMalloc(&d_output_forward_multiple_nn_sum_pointers, number_networks * sizeof(float*));
	for (int i = 0; i < number_networks; i++) { hd_output_forward_multiple_nn_sum_pointers[i] = d_output_forward_multiple_nn_sum + i*output_size; }
	hipMemcpy(d_output_forward_multiple_nn_sum_pointers, hd_output_forward_multiple_nn_sum_pointers, number_networks * sizeof(float*), hipMemcpyHostToDevice);
	delete hd_output_forward_multiple_nn_sum_pointers;

	int first_max = max(max_layer_size, input_size);
	int second_max = 0;
	for (int i = 0; i < number_layers; i++) {
		if (max(second_max, layers[i]->getSize()) < first_max) { second_max = max(second_max, layers[i]->getSize()); }
	}

	//Cublas warmup
	productoMatricesDevice(handle, d_auxiliar_expand_reduce_matrix, layers[number_layers - 1]->getDeviceForward(), d_output_forward_multiple_nn_sum, 1, number_networks, output_size);

	hipDeviceSynchronize();
}

void Network::initWeightBiasValues() {
	hiprandGenerator_t curandGenerator;
	hiprandCreateGenerator(&curandGenerator, HIPRAND_RNG_PSEUDO_MT19937);
	hiprandSetGeneratorOrdering(curandGenerator, HIPRAND_ORDERING_PSEUDO_BEST);
	hipDeviceSynchronize();
	for (int i = 0; i < number_layers; i++) {
		layers[i]->initWeightBiasValues(curandGenerator);
	}
	hipDeviceSynchronize();
	hiprandDestroyGenerator(curandGenerator);
}

const void Network::copyInputOutputTrain(int num_examples, float* input_data, float* output_data) {
	if (num_examples <= max_input_number_examples) {
		hipMemcpyAsync(h_pinned_input_matrix, input_data, num_examples * input_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
		hipMemcpyAsync(h_pinned_output_matrix, output_data, num_examples * output_size * sizeof(float), hipMemcpyHostToHost, stream_transferencia_output);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix, h_pinned_input_matrix, num_examples * input_size * sizeof(float), hipMemcpyHostToDevice, stream_principal);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix + d_pinned_output_offset, h_pinned_output_matrix, num_examples * output_size * sizeof(float), hipMemcpyHostToDevice, stream_transferencia_output);
		//hipStreamSynchronize(stream_principal);
		//hipStreamSynchronize(stream_transferencia_output);
		//para hacer el backward, esperar� a que ambas transferencias hayan terminado
		/*
		//input y output se copian bien, deber�an de llegar a tiempo sin necesidad de sincronizaci�n
		float* inputarr = new float[input_size * num_examples];
		float* outputarr = new float[output_size * num_examples];
		hipMemcpy(inputarr, d_pinned_input_output_auxiliar_matrix, input_size * num_examples * sizeof(float), hipMemcpyDeviceToHost);
		hipMemcpy(outputarr, d_pinned_input_output_auxiliar_matrix + d_pinned_output_offset, output_size * num_examples * sizeof(float), hipMemcpyDeviceToHost);
		imprimirMatrizPorPantalla("input", inputarr, num_examples, input_size);
		imprimirMatrizPorPantalla("output", outputarr, num_examples, output_size);
		*/
	}
	else {
		printf("\nCannot copy input and output, more examples than max number of examples defined in initForward");
	}
}

const void Network::forward(int num_examples, float* input_data, float* output_pointer_dest) {
	if (num_examples <= max_input_number_examples) {
		hipMemcpyAsync(h_pinned_input_matrix, input_data, num_examples * input_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
		hipMemcpyAsync(d_pinned_input_output_auxiliar_matrix, h_pinned_input_matrix, num_examples * input_size * sizeof(float), hipMemcpyHostToDevice, stream_principal);
		layers[0]->forward(stream_principal, d_input_pointers, num_examples);
		for (int i = 1; i < number_layers; i++) {
			layers[i]->forward(stream_principal, layers[i-1], num_examples);
		}

		if (number_networks == 1) {
			hipMemcpyAsync(h_pinned_output_matrix, layers[number_layers-1]->getDeviceForward(), num_examples * output_size * sizeof(float), hipMemcpyDeviceToHost, stream_principal);
			hipMemcpyAsync(output_pointer_dest, h_pinned_output_matrix, num_examples * output_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
		} else {
			if (max_input_number_examples == 1) {
				productoMatricesDevice(handle, d_auxiliar_expand_reduce_matrix, layers[number_layers - 1]->getDeviceForward(), d_output_forward_multiple_nn_sum, 1, number_networks, output_size);
				
				managedMultiplyAllElementsByConstant(stream_principal, max_num_threads, nextFourMultiple(output_size), d_output_forward_multiple_nn_sum, 1 / (float)number_networks);
				//multiplyAllElementsByConstantVectorial << < (int)ceil(nextFourMultiple(num_examples * output_size) /(float)(max_num_threads*4)), min(max_num_threads, nextFourMultiple(num_examples * output_size) / 4), 0, stream_principal >> > (d_output_forward_multiple_nn_sum, 1 / (float)number_networks);
				hipMemcpyAsync(h_pinned_output_matrix, d_output_forward_multiple_nn_sum, num_examples * output_size * sizeof(float), hipMemcpyDeviceToHost, stream_principal);
				hipMemcpyAsync(output_pointer_dest, h_pinned_output_matrix, num_examples * output_size * sizeof(float), hipMemcpyHostToHost, stream_principal);
			} else {
				//habr� que hacer el sumatorio de todas las matrices al de todas las redes, y multiplicarles 1/numero_redes
			}
		}
		hipStreamSynchronize(stream_principal);
	} else {
		printf("\nCannot make forward, more examples than max number of examples defined in initForward");
	}
}

const void Network::forwardTrain(int num_examples) {
	layers[0]->forward(stream_principal, d_input_pointers, num_examples);
	for (int i = 1; i < number_layers; i++) {
		layers[i]->forward(stream_principal, layers[i - 1], num_examples);
	}
	hipStreamSynchronize(stream_principal);
	hipStreamSynchronize(stream_transferencia_output);
}

//first batch_id = 0
const void Network::forwardTrain(int num_examples, int batch_size, float** d_input_pointers) {
	layers[0]->forward(stream_principal, d_input_pointers, batch_size);
	for (int i = 1; i < number_layers; i++) {
		layers[i]->forward(stream_principal, layers[i - 1], batch_size);
	}
	hipStreamSynchronize(stream_principal);
	hipStreamSynchronize(stream_transferencia_output);
}

float* Network::trainGetCostFunctionAndCalculateLossFunction(int num_examples) {
	int* pos = new int[number_networks];
	for (int i = 0; i < number_networks; i++) { pos[i] = 0; }
	return trainGetCostFunctionAndCalculateLossFunction(num_examples, num_examples, pos);
}

//first batch_id = 0
float* Network::trainGetCostFunctionAndCalculateLossFunction(int num_examples, int batch_size, int* batch_ids) {
	if (batch_size <= max_input_number_examples) {
		if (num_examples % batch_size == 0) {
			int num_elems_batch = batch_size * input_size;
			float** ptrs = new float* [number_networks];
			for (int i = 0; i < number_networks; i++) { ptrs[i] = d_pinned_input_output_auxiliar_matrix + (batch_ids[i] * num_elems_batch); }
			hipMemcpy(d_input_pointers, ptrs, number_networks * sizeof(float*), hipMemcpyHostToDevice);
			forwardTrain(num_examples, batch_size, d_input_pointers);
			delete ptrs;

			num_elems_batch = batch_size * output_size;
			
			/*
			float* matriz_Cost2 = new float[num_elems_batch];
			for (int i = 0; i < number_networks; i++) {
				hipMemcpy(matriz_Cost, layers[number_layers - 1]->getDeviceForward() + (i * num_elems_batch), num_elems_batch * sizeof(float), hipMemcpyDeviceToHost);
				imprimirMatrizPorPantalla("XD:", matriz_Cost, batch_size, output_size);
				hipMemcpy(matriz_Cost, d_pinned_input_output_auxiliar_matrix + d_pinned_output_offset + (batch_ids[i] * num_elems_batch), num_elems_batch * sizeof(float), hipMemcpyDeviceToHost);
				imprimirMatrizPorPantalla("XD2:", matriz_Cost, batch_size, output_size);
			}
			delete matriz_Cost2;
			*/

			//apply cost function
			for (int i = 0; i < number_networks; i++) {
				managedApplyLossFunction(stream_principal, max_num_threads, num_elems_batch,
					layers[number_layers - 1]->getDeviceForward() + (i * num_elems_batch),
					d_pinned_input_output_auxiliar_matrix + d_pinned_output_offset + (batch_ids[i] * num_elems_batch),
					d_auxiliar_matrix_loss_function_error_backprop + (i * num_elems_batch),
					loss_function
				);
			}

			//obtain cost function
			float* cost_function_result = new float[number_networks];
			float* d_res = 0;
			hipMalloc(&d_res,number_networks * sizeof(float));
			productoMatricesBatchDevice(handle, layers[number_layers-1]->getAuxiliarExpandReduceMatrixPointers(), layers[number_layers - 1]->getDeviceAuxiliarErrorForwardLayerPointers(), d_output_forward_multiple_nn_sum_pointers, 1, batch_size, output_size, number_networks);
			
			managedMultiplyAllElementsByConstant(stream_principal, max_num_threads, output_size * number_networks, d_output_forward_multiple_nn_sum, 1 / (float)(batch_size));

			/*multiplyAllElementsByConstantVectorial << < (int)ceil(output_size * number_networks / 4), min(max_num_threads, (int)(output_size * number_networks / 4)), 0, stream_principal >> > (d_output_forward_multiple_nn_sum, 1 / (float)(batch_size));
			if ((output_size * number_networks) % 4 != 0) {
				multiplyAllElementsByConstantScalar << < 1, (output_size * number_networks) % 4, 0, stream_principal >> > (d_output_forward_multiple_nn_sum + (((output_size * number_networks) / 4) * 4), 1 / (float)(batch_size));
			}
			*/
			productoMatricesDevice(handle, d_auxiliar_expand_reduce_matrix, d_output_forward_multiple_nn_sum, d_res, 1, output_size, number_networks);
			hipMemcpy(cost_function_result, d_res, number_networks*sizeof(float), hipMemcpyDeviceToHost);
			for(int i = 0; i < number_networks;i++){ cost_function_result[i] = cost_function_result[i] / (float) output_size; }
			/*for (int i = 0; i < number_networks; i++) {
				productoMatricesDevice(handle, d_auxiliar_expand_reduce_matrix, d_auxiliar_matrix_loss_function_error_backprop + i*batch_size, d_output_forward_multiple_nn_sum, 1, batch_size, output_size);
				multiplyAllElementsByConstantVectorial << < (int)ceil(output_size / 4), min(max_num_threads, (int)(output_size / 4)), 0, stream_principal >> > (d_output_forward_multiple_nn_sum, 1 / (float)(batch_size));
				if (output_size % 4 != 0) {
					multiplyAllElementsByConstantScalar << < 1, output_size % 4, 0, stream_principal >> > (d_output_forward_multiple_nn_sum + ((output_size / 4) * 4), 1 / (float)(batch_size));
				}

				productoMatricesDevice(handle, d_auxiliar_expand_reduce_matrix, d_output_forward_multiple_nn_sum, d_res, 1, output_size, 1);
				hipMemcpy(&cost_function_result[i], d_res, sizeof(float), hipMemcpyDeviceToHost);
				cost_function_result[i] = cost_function_result[i] / (float)output_size;
			}*/
			hipFree(d_res);

			/*
			float* matriz_Cost = new float[num_elems_batch * number_networks];
			hipMemcpy(matriz_Cost, d_auxiliar_matrix_loss_function_error_backprop, num_elems_batch * number_networks * sizeof(float), hipMemcpyDeviceToHost);
			imprimirMatrizPorPantalla("Error de coste:", matriz_Cost, batch_size* number_networks, output_size);
			delete matriz_Cost;
			*/

			/*
			float* mirarCost = new float[number_networks*output_size];
			hipMemcpy(mirarCost, d_output_forward_multiple_nn_sum, number_networks * output_size * sizeof(float), hipMemcpyDeviceToHost);
			imprimirMatrizPorPantalla("Error de coste sum:", mirarCost, number_networks, output_size);
			delete mirarCost;
			*/

			//apply loss function
			for (int i = 0; i < number_networks; i++) {
				managedApplyLossFunction(stream_principal, max_num_threads, num_elems_batch,
					layers[number_layers - 1]->getDeviceForward() + (i * num_elems_batch),
					d_pinned_input_output_auxiliar_matrix + d_pinned_output_offset + (batch_ids[i] * num_elems_batch),
					d_auxiliar_matrix_loss_function_error_backprop + (i * num_elems_batch),
					derivative_loss_function
				);
			}

			/*
			float* matriz_Cost = new float[num_elems_batch * number_networks];
			hipMemcpy(matriz_Cost, d_auxiliar_matrix_loss_function_error_backprop, num_elems_batch * number_networks * sizeof(float), hipMemcpyDeviceToHost);
			imprimirMatrizPorPantalla("Derivada Error de coste:", matriz_Cost, batch_size * number_networks, output_size);
			delete matriz_Cost;
			*/

			return cost_function_result;
		} else {
			printf("\nwhen batch forwardTrain, num_examples % batch_size must be 0");
		}
	} else {
		printf("\nCannot make forward, more examples than max number of examples defined in initForward");
	}
	return NULL;
}

float* Network::backwardPhase(int num_examples, int batch_size, int* batch_ids) {
	if (batch_size <= max_input_number_examples) {
		if (num_examples % batch_size == 0) {
			float* cost_function = trainGetCostFunctionAndCalculateLossFunction(num_examples, batch_size, batch_ids);
			for (int i = number_layers - 1; i > 0; i--) {
				layers[i]->backward(stream_principal, layers[i - 1], batch_size);
			}
			layers[0]->backward(stream_principal, d_input_pointers, batch_size);
			return cost_function;
		} else {
			printf("\nwhen batch forwardTrain, num_examples % batch_size must be 0");
		}
	} else {
		printf("\nCannot make forward, more examples than max number of examples defined in initForward");
	}
	return NULL;
}

void Network::applyVGradSGD(float lrate) {
	for (int i = 0; i < number_layers; i++) {
		layers[i]->applyGradientSGD(stream_principal, lrate);
	}
}

void Network::finalizeForward() {
	hipblasSetStream(handle, 0);
	hipStreamDestroy(stream_principal);
	hipStreamDestroy(stream_transferencia_output);
	hipFree(d_pinned_input_output_auxiliar_matrix);
	hipFree(d_input_pointers);
	hipHostFree(h_pinned_input_matrix);
	hipHostFree(h_pinned_output_matrix);
	
	for (int i = 0; i < number_layers; i++) {
		layers[i]->setNumberInputExamples(0);
		layers[i]->setAuxiliarExpandReduceMatrix(NULL);
		layers[i]->freeForwardMemory();
		layers[i]->freeBackwardMemory();
		layers[i]->setIsTraining(false);
	}
	hipFree(d_auxiliar_expand_reduce_matrix);

	hipFree(d_output_forward_multiple_nn_sum);
	if (d_output_forward_multiple_nn_sum_pointers != NULL) { hipFree(d_output_forward_multiple_nn_sum_pointers);  d_output_forward_multiple_nn_sum_pointers = NULL; }

	if (d_auxiliar_matrix_loss_function_error_backprop != NULL) { hipFree(d_auxiliar_matrix_loss_function_error_backprop); d_auxiliar_matrix_loss_function_error_backprop = NULL; }
	if (d_auxiliar2_matrix_loss_function_error_backprop != NULL) { hipFree(d_auxiliar2_matrix_loss_function_error_backprop); d_auxiliar2_matrix_loss_function_error_backprop = NULL; }

	hipDeviceSynchronize();
	max_batch_size = 0;
	max_input_number_examples = 0;
}