#include "hip/hip_runtime.h"
#include "CUDAKernels.cuh"

void manageCUDAError(hipError_t status, char* description) {
    if (status != hipSuccess) {
        fprintf(stderr, "Error de CUDA %s: %s\n", description, hipGetErrorString(status));
        exit(EXIT_FAILURE);
    }
}

const void productoMatricesDevice(hipblasHandle_t handle, const float* a, const float* b, float* c, int m, int k, int n) {
    hipblasSgemm_64(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, b, n, a, k, &beta_nosum, c, n);
}

const void productoMatricesBatchDevice(hipblasHandle_t handle, float** a, float** b, float** c, int m, int k, int n, int num_matr) {
    hipblasSgemmBatched_64(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, b, n, a, k, &beta_nosum, c, n, num_matr);
}

__global__ void applyFunctionVectorial(float* arr, func_t func) {
    //https://forums.developer.nvidia.com/t/the-float-and-float4-types-in-cuda/65061
    float4 val = reinterpret_cast<float4*>(arr)[blockIdx.x * blockDim.x + threadIdx.x];
    val.x = func(val.x);
    val.y = func(val.y);
    val.z = func(val.z);
    val.w = func(val.w);
    reinterpret_cast<float4*>(arr)[blockIdx.x * blockDim.x + threadIdx.x] = val;
}