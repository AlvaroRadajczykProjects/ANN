#include "hip/hip_runtime.h"
#include "Layer.cuh"

#define N 10

using namespace std;

int nextFourMultiple(int val) {
    if (val % 4 == 0) { return val; }
    else { return val + (4 - (val % 4)); }
}

Layer::Layer(int sz, func_t dev_act_func, func_t dev_act_der_func) {
	size = sz;
    activation_function = dev_act_func;
    activation_derivative_function = dev_act_der_func;

    /*
    hipError_t cudaStatus;

    float* p = 0;
    hipMalloc((void**)&p, N * sizeof(float));

    float* h_p = new float[N];
    for (int i = 0; i < N; i++) { h_p[i] = i - 9; }

    imprimirVectorPorPantalla("antes", h_p, 0, N);

    hipMemcpy(p, h_p, N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time;

    applyFunctionVectorial << < (int)ceil(N / (float)(1024 * 4)), 1024 >> > (p, activation_function);

    hipDeviceSynchronize();

    hipMemcpy(h_p, p, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    imprimirVectorPorPantalla("despues", h_p, 0, N);

    hipFree(p);
    free(h_p);
    */
}

Layer::~Layer() {
    handle = NULL;
}

void Layer::showInfo() {
    printf("\n\t\tInput size: %d", input_size);
    printf("\n\t\tSize: %d", size);
    printf("\n\t\tNumber of networks: %d", number_networks);
    printf("\n\t\tFirst layer?: %s", is_first_layer ? "Yes" : "No");
    printf("\n\t\tIs training?: %s", is_training ? "Yes" : "No");
    printf("\n");
}

void Layer::showWeightBias() {
    for (int i = 0; i < number_networks; i++) {
        float* h_weight_m = new float[input_size * size];
        float* h_bias_v = new float[size];
        hipMemcpy(h_weight_m, hd_weight_matrices_pointers[i], input_size * size * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_bias_v, hd_bias_vectors_pointers[i], size * sizeof(float), hipMemcpyDeviceToHost);
        printf("\n\tNetwork %d:", i);
        imprimirMatrizPorPantalla("\n\t\tbias:", h_bias_v, 1, size);
        imprimirMatrizPorPantalla("\n\t\tweight:", h_weight_m, input_size, size);
        delete h_weight_m;
        delete h_bias_v;
    }
}

void Layer::showAuxiliarExpandReduce() {
    for (int i = 0; i < number_networks; i++) {
        float* h_auxiliar_expand_reduce_matrix = new float[number_input_examples];
        hipMemcpy(h_auxiliar_expand_reduce_matrix, hd_expand_reduce_matrix_pointers[i], number_input_examples * sizeof(float), hipMemcpyDeviceToHost);
        printf("\n\tNetwork %d:", i);
        imprimirMatrizPorPantalla("\n\t\tauxiliar:", h_auxiliar_expand_reduce_matrix, 1, size);
        delete h_auxiliar_expand_reduce_matrix;
    }
}

void Layer::showForward() {
    for (int i = 0; i < number_networks; i++) {
        float* h_forward = new float[number_input_examples * size];
        hipMemcpy(h_forward, hd_forward_pointers[i], number_input_examples * size * sizeof(float), hipMemcpyDeviceToHost);
        printf("\n\tNetwork %d:", i);
        imprimirMatrizPorPantalla("\n\t\tforward matrix:", h_forward, number_input_examples, size);
        delete h_forward;
    }
}

int Layer::getSize() {
    return size;
}

float** Layer::getDeviceForwardPointers() {
    return d_forward_pointers;
}

void Layer::setMaxNumThreads(int set) {
    max_num_threads = set;
}

void Layer::setInputSize(int is) {
    input_size = is;
}

void Layer::setNumberInputExamples(int set) {
    number_input_examples = set;
}

void Layer::setAuxiliarExpandReduceMatrix(float* set) {
    d_auxiliar_expand_reduce_matrix = set;
    if (set != NULL) {
        hd_expand_reduce_matrix_pointers = new float* [number_networks];
        hipMalloc(&d_expand_reduce_matrix_pointers, number_networks * sizeof(float*));
        for (int i = 0; i < number_networks; i++) { hd_expand_reduce_matrix_pointers[i] = d_auxiliar_expand_reduce_matrix; }
        hipMemcpy(d_expand_reduce_matrix_pointers, hd_expand_reduce_matrix_pointers, number_networks * sizeof(float*), hipMemcpyHostToDevice);
    } else {
        hipFree(d_expand_reduce_matrix_pointers);
        delete hd_expand_reduce_matrix_pointers;
    }
}

void Layer::setNumberNetworks(int nn) {
    number_networks = nn;
}

void Layer::setIsFirstLayer(bool set) {
    is_first_layer = set;
}

void Layer::setIsTraining(bool set) {
    is_training = set;
}

void Layer::setCublasHandle(hipblasHandle_t* h) {
    handle = h;
}

void Layer::forward(hipStream_t stream, float** d_input_pointers) {
    productoMatricesBatchDevice(*handle, d_expand_reduce_matrix_pointers, d_bias_vectors_pointers, d_forward_pointers, number_input_examples, 1, size, number_networks);
    productoMatricesBatchDeviceSumC(*handle, d_input_pointers, d_weight_matrices_pointers, d_forward_pointers, number_input_examples, input_size, size, number_networks);
    applyFunctionVectorial <<< num_blocks_needed_apply_function, num_threads_needed_apply_function, 0, stream >> > (d_forward, activation_function);
}

void Layer::forward(hipStream_t stream, Layer* previous_layer) {
    productoMatricesBatchDevice(*handle, d_expand_reduce_matrix_pointers, d_bias_vectors_pointers, d_forward_pointers, number_input_examples, 1, size, number_networks);
    productoMatricesBatchDeviceSumC(*handle, previous_layer->getDeviceForwardPointers(), d_weight_matrices_pointers, d_forward_pointers, number_input_examples, input_size, size, number_networks);
    applyFunctionVectorial << < num_blocks_needed_apply_function, num_threads_needed_apply_function, 0, stream >> > (d_forward, activation_function);
}

void Layer::allocWeightMatricesMemory() {
    if (input_size > 0 && size > 0 && number_networks > 0) {
        hipMalloc( &d_array_weight_matrix, input_size * size * number_networks * sizeof(float));
        hipMalloc( &d_array_bias_vector, size * number_networks * sizeof(float));
        hd_weight_matrices_pointers = new float* [number_networks];
        hd_bias_vectors_pointers = new float* [number_networks];
        hipMalloc(&d_weight_matrices_pointers, number_networks * sizeof(float*));
        hipMalloc(&d_bias_vectors_pointers, number_networks * sizeof(float*));
        for (int i = 0; i < number_networks; i++) {
            hd_weight_matrices_pointers[i] = d_array_weight_matrix + i*(input_size * size);
            hd_bias_vectors_pointers[i] = d_array_bias_vector + i * (size);
        }
        hipMemcpy(d_weight_matrices_pointers, hd_weight_matrices_pointers, number_networks * sizeof(float*), hipMemcpyHostToDevice);
        hipMemcpy(d_bias_vectors_pointers, hd_bias_vectors_pointers, number_networks * sizeof(float*), hipMemcpyHostToDevice);
    }
}

void Layer::freeWeightMatricesMemory() {
    if (input_size > 0 && size > 0 && number_networks > 0) {
        hipFree(d_array_weight_matrix); d_array_weight_matrix = NULL;
        hipFree(d_array_bias_vector); d_array_bias_vector = NULL;
        delete hd_weight_matrices_pointers;  hd_weight_matrices_pointers = NULL;
        delete hd_bias_vectors_pointers; hd_bias_vectors_pointers = NULL;
        hipFree(d_weight_matrices_pointers); d_weight_matrices_pointers = NULL;
        hipFree(d_bias_vectors_pointers); d_bias_vectors_pointers = NULL;
    }
    input_size = 0;
    size = 0;
    number_networks = 0;
}

void Layer::allocForwardMemory() {
    if (input_size > 0 && size > 0 && number_networks > 0 && number_input_examples > 0) {
        num_blocks_needed_apply_function = (int)ceil((size * number_input_examples) / (float)(max_num_threads * 4));
        num_threads_needed_apply_function = min(max_num_threads, number_input_examples * size * 4);

        hipMalloc(&d_forward, nextFourMultiple( number_input_examples * size * number_networks ) * sizeof(float));
        hd_forward_pointers = new float* [number_networks];
        hipMalloc(&d_forward_pointers, number_networks * sizeof(float*));
        for (int i = 0; i < number_networks; i++) {
            hd_forward_pointers[i] = d_forward + ( i * number_input_examples * size);
        }
        hipMemcpy(d_forward_pointers, hd_forward_pointers, number_networks * sizeof(float*), hipMemcpyHostToDevice);
    }
}

void Layer::freeForwardMemory() {
    if (input_size > 0 && size > 0 && number_networks > 0 && number_input_examples > 0) {
        num_blocks_needed_apply_function = 0;
        num_threads_needed_apply_function = 0;

        hipFree(d_forward); d_forward = NULL;
        delete hd_forward_pointers; hd_forward_pointers = NULL;
        hipFree(d_forward_pointers); d_forward_pointers = NULL;
    }
    number_input_examples = 0;
}

void Layer::copyWeightBias(float* h_weight, float* h_bias) {
    if (input_size > 0 && size > 0 && number_networks > 0) {
        hipMemcpy(d_array_weight_matrix, h_weight, input_size * size * number_networks * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_array_bias_vector, h_bias, size * number_networks * sizeof(float), hipMemcpyHostToDevice);
    }
}