#include "hip/hip_runtime.h"
#include "Layer.cuh"

#define N 10

Layer::Layer(int sz, func_t dev_act_func, func_t dev_act_der_func) {
	size = sz;
    activation_function = dev_act_func;
    activation_derivative_function = dev_act_der_func;

    /*
    hipError_t cudaStatus;

    float* p = 0;
    hipMalloc((void**)&p, N * sizeof(float));

    float* h_p = new float[N];
    for (int i = 0; i < N; i++) { h_p[i] = i - 9; }

    imprimirVectorPorPantalla("antes", h_p, 0, N);

    hipMemcpy(p, h_p, N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time;

    applyFunctionVectorial << < (int)ceil(N / (float)(1024 * 4)), 1024 >> > (p, activation_function);

    hipDeviceSynchronize();

    hipMemcpy(h_p, p, N * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    imprimirVectorPorPantalla("despues", h_p, 0, N);

    hipFree(p);
    free(h_p);
    */
}

Layer::~Layer() {
    handle = NULL;
}

void Layer::showInfo() {
    printf("\n\t\tInput size: %d", input_size);
    printf("\n\t\tSize: %d", size);
    printf("\n\t\tNumber of networks: %d", number_networks);
    printf("\n");
}

void Layer::showWeightBias() {
    float* h_weight_m = new float[input_size * size * number_networks];
    float* h_bias_v = new float[size * number_networks];
    hipMemcpy(h_weight_m, d_array_weight_matrix, input_size * size * number_networks * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_bias_v, d_array_bias_vector, size * number_networks * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < number_networks; i++) {
        printf("\n\tNetwork %d:", i);
        imprimirMatrizPorPantalla("\n\t\tbias:", h_bias_v + (i * size), 1, size);
        imprimirMatrizPorPantalla("\n\t\tweight:", h_weight_m + (i * input_size * size), input_size, size);
    }
    delete h_weight_m;
    delete h_bias_v;
}

int Layer::getSize() {
    return size;
}

void Layer::setInputSize(int is) {
    input_size = is;
}

void Layer::setNumberNetworks(int nn) {
    number_networks = nn;
}

void Layer::setCublasHandle(hipblasHandle_t* h) {
    handle = h;
}

void Layer::allocMemory() {
    if (input_size > 0 && size > 0 && number_networks > 0) {
        hipMalloc( &d_array_weight_matrix, input_size * size * number_networks * sizeof(float));
        hipMalloc( &d_array_bias_vector, size * number_networks * sizeof(float));
    }
}

void Layer::freeMemory() {
    if (input_size > 0 && size > 0 && number_networks > 0) {
        hipFree(d_array_weight_matrix); d_array_weight_matrix = NULL;
        hipFree(d_array_bias_vector); d_array_bias_vector = NULL;
    }
    input_size = 0;
    size = 0;
    number_networks = 0;
}

void Layer::copyWeightBias(float* h_weight, float* h_bias) {
    if (input_size > 0 && size > 0 && number_networks > 0) {
        hipMemcpy(d_array_weight_matrix, h_weight, input_size * size * number_networks * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_array_bias_vector, h_bias, size * number_networks * sizeof(float), hipMemcpyHostToDevice);
    }
}