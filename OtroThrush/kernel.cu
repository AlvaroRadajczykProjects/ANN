#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

void imprimirVectorPorPantalla(char* texto_mostrar, float vector[], int inicio, int fin) {
    printf("\n%s [ ", texto_mostrar);
    for (int i = inicio; i < fin; i++) {
        printf("%.8f", vector[i]);
        if (i < fin - 1) { printf(","); }
        printf(" ");
    }
    printf("]");
}

const size_t ds = 1024;
const size_t rows = 8192;
const int nTPB = 1024;

__constant__ float vector[ds]; //16364

__global__ void k(float* dout) {
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    float my_val;
    if (idx < ds) {
        my_val = vector[idx];
        for (size_t i = 0; i < rows; i++) {
            dout[idx] = my_val;
            idx += ds;
        }
    }
}

int main() {


    float* d_in = 0;
    float* d_out = 0;
    float h_in[ds];
    float* h_out = new float[ds * rows];
    for (int i = 0; i < ds; i++) h_in[i] = i+1;

    hipMalloc(&d_out, ds * rows * sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(vector), &h_in, sizeof(float) * ds);

    //hipMemcpy(d_in, h_in, ds * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float time;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    for(int i = 0; i < 16; i++)
        k << <(ds + nTPB - 1) / nTPB, nTPB >> > (d_out);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("\n\nExecution time: %f ms\n\n", time);

    hipMemcpy(h_out, d_out, ds * rows * sizeof(float), hipMemcpyDeviceToHost);
    
    imprimirVectorPorPantalla("", h_out, 8192-1024, 8192);
    
}